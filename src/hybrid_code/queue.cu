//
// Created by Khurram Javed on 2021-04-11.
//
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <exception>
#include <stdexcept>

//
struct queue_elem {
    float value;
    queue_elem *next_pointer;
    queue_elem *prev_pointer;

    queue_elem() {
        next_pointer = nullptr;
        prev_pointer = nullptr;
    }
};

class base_queue {
protected:
    queue_elem *back;
    queue_elem *front;
public:
    base_queue() {
        back = nullptr;
        front = nullptr;
    }

    virtual void add_elem(float value) {
    }

    virtual float pop_front() {
        return 0;
    }

    void print_queue() {
        queue_elem *temp = back;
        while (temp != nullptr) {
            std::cout << temp->value << ",";
            temp = temp->next_pointer;
        }
        std::cout << "\n";
    }
};

class device_queue : public base_queue {
public:
    ~device_queue() {
        queue_elem *temp = back;
        while (temp != nullptr) {
            queue_elem *temp_for_free = temp;
            temp = temp->next_pointer;
            hipFree(temp_for_free);
        }
    }

//
    device_queue() = default;

    void add_elem(float value) override {
        void *test;
        hipMallocManaged(&test, sizeof(queue_elem));
//        queue_elem *new_elem = new queue_elem();
        auto *new_elem = static_cast<queue_elem *>(test);
        new_elem->next_pointer = nullptr;
        new_elem->value = value;
        if (back == nullptr) {
            if (front != nullptr) {
                std::cout << "Impossible situation \n";
                exit(1);
            }
            new_elem->prev_pointer = nullptr;
            back = new_elem;
            front = new_elem;
        } else {
            new_elem->prev_pointer = nullptr;
            new_elem->next_pointer = back;
            back->prev_pointer = new_elem;
            back = new_elem;
        }
    }

    float pop_front() override {
        if (front == nullptr) {
            std::cout << "Can't pop element from an empty device_queue\n";
            exit(1);
        }
        float return_val = front->value;
        if (front->prev_pointer != nullptr) {
            front->prev_pointer->next_pointer = nullptr;
        }
        queue_elem *temp = front->prev_pointer;
        hipFree(front);
        front = temp;
        return return_val;
    }
};


class host_queue : public base_queue {
public:
    ~host_queue() {
        queue_elem *temp = back;
        while (temp != nullptr) {
            queue_elem *temp_for_free = temp;
            temp = temp->next_pointer;
            delete temp_for_free;
        }
    }

    host_queue() = default;

    void add_elem(float value) override {
        auto *new_elem = new queue_elem;
        new_elem->next_pointer = nullptr;
        new_elem->value = value;
        if (back == nullptr) {
            if (front != nullptr) {
                std::cout << "Impossible situation \n";
                exit(1);
            }
            new_elem->prev_pointer = nullptr;
            back = new_elem;
            front = new_elem;
        } else {
            new_elem->prev_pointer = nullptr;
            new_elem->next_pointer = back;
            back->prev_pointer = new_elem;
            back = new_elem;
        }
    }

    float pop_front() override {
        if (front == nullptr) {
            std::cout << "Can't pop element from an empty device_queue\n";
            exit(1);
        }
        float return_val = front->value;
        if (front->prev_pointer != nullptr) {
            front->prev_pointer->next_pointer = nullptr;
        }
        queue_elem *temp = front->prev_pointer;
        delete front;
        front = temp;
        return return_val;
    }
};
