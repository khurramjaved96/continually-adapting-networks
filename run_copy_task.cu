#include "hip/hip_runtime.h"
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include "include/utils.h"
#include <map>
#include <string>

#include "include/neural_networks/networks/test_network.h"
#include "include/neural_networks/neural_network.h"
#include "include/experiment/Experiment.h"
#include "include/neural_networks/utils.h"
#include "include/experiment/Metric.h"
#include "src/hybrid_code/queue.cu"
#include "include/environments/copy_task.h"
#include "include/animal_learning/tracecondioning.h"
#include "include/neural_networks/networks/test_network.h"


int main(int argc, char *argv[]) {
//    std::string default_config = "--name test --width 10 --seed 0 --steps 100 --run 0 --step_size 0.0001 --num_layers 5 --data_driven_initialization false --randomize_sequence_length true --add_features false --sparsity 98 --sequence_gap 20"
//    __builtin_trap();
    std::cout << "Program started \n";
    Experiment exp = Experiment(argc, argv);
    std::cout << "Experiment object created \n";
    int width = exp.get_int_param("width");

    Metric synapses_metric = Metric(exp.database_name, "error_table",
                                    std::vector<std::string>{"step", "datatime", "seq_length", "run", "error", "accuracy"},
                                    std::vector<std::string>{"int", "int", "int", "int", "real", "real"},
                                    std::vector<std::string>{"step",  "run" });
    Metric run_state_metric = Metric(exp.database_name, "state_table",
                                     std::vector<std::string>{"run", "state", "state_comments"},
                                     std::vector<std::string>{"int", "MEDIUMTEXT", "MEDIUMTEXT"},
                                     std::vector<std::string>{"run"});
    Metric observations_metric = Metric(exp.database_name, "obs_table",
                                        std::vector<std::string>{"run", "step", "inp_start_flag", "inp_end_flag", "stml_seq", "target", "pred", "L", "seq_len", "data_timestep"},
                                        std::vector<std::string>{"int", "int", "real", "real", "real", "real", "real", "int", "int", "int"},
                                        std::vector<std::string>{"run", "step"});
    Metric graph_state = Metric(exp.database_name, "graph",
                                std::vector<std::string>{"step", "run", "graph_data"},
                                std::vector<std::string>{"int", "int", "MEDIUMTEXT"},
                                std::vector<std::string>{"step", "run"});

    CustomNetwork my_network = CustomNetwork(exp.get_float_param("step_size"),
                                             exp.get_int_param("width"),
                                             exp.get_int_param("num_layers"),
                                             exp.get_int_param("sparsity"),
                                             exp.get_int_param("seed"));

    CopyTask env = CopyTask(exp.get_int_param("seed"),
                            exp.get_bool_param("randomize_sequence_length"),
                            exp.get_int_param("sequence_gap"));
    //get a sequence of data for data-driven initialization
    if (exp.get_bool_param("data_driven_initialization")){
        std::vector<std::vector<float>> input_batch;
        input_batch.reserve(500);
        for(int temp=0; temp<500; temp++)
            input_batch.push_back(env.step(1));
        my_network.initialize_network(input_batch);
        env.reset();
    }

    std::cout << "Total synapses in the network " << my_network.get_total_synapses() << std::endl;
    auto start = std::chrono::steady_clock::now();

    float running_error = -1;
    float running_accuracy = -1;
    float target = 0;
    float prediction = 0;
    float last_err = 1;
    float target_old = 0;
    int current_seq_length = 1;
    std::string state = "finished";
    std::string state_comments = "";
    std::vector<std::vector<std::string>> error_logger;
    std::vector<std::vector<std::string>> state_logger;
    std::vector<std::vector<std::string>> graph_logger;

    std::cout << "Flag Bit \t Pred Bit \t Target \t Pred \t Seq_len \t Datatime" << std::endl;
    for (int counter = 0; counter < exp.get_int_param("steps"); counter++) {

        auto state_current = env.step(last_err);
//        print_vector(state_current);
        my_network.set_input_values(state_current);
        my_network.step();
        prediction = my_network.read_output_values()[0];
        if(isnan(prediction)){
          state = "killed";
          state_comments = "nan_prediction";
          break;
        }

        target_old = target;
        target = env.get_target();
        if(counter > 0)
            my_network.introduce_targets(std::vector<float>{target_old});

        float error = (prediction - target)*(prediction - target);
        last_err = error;
        float accuracy;
        (prediction > 0.5 && target == 1) || (prediction < 0.5 && target == 0) ? accuracy=1 : accuracy=0;
        if (running_error == -1){
            running_error = error;
            running_accuracy = accuracy;
        }
        else{
            running_error = running_error * 0.999 + 0.001 *error;
            running_accuracy = running_accuracy * 0.999 + 0.001 *accuracy;
        }



        if(env.get_L() > current_seq_length || counter % 300 == 0)
        {
            std::vector<std::string> error_vec;
            error_vec.push_back(std::to_string(counter));
            error_vec.push_back(std::to_string(env.get_data_timestep()));
            error_vec.push_back(std::to_string(env.get_L()));
            error_vec.push_back(std::to_string(exp.get_int_param("run")));
            error_vec.push_back(std::to_string(running_error));
            error_vec.push_back(std::to_string(running_accuracy));
            error_logger.push_back(error_vec);
            current_seq_length = env.get_L();
        }


        std::vector<float> cur_state = env.get_state();

        if(counter < 1000 || counter > exp.get_int_param("steps") - 1000)
        {
            std::vector<std::string> state_vec;
            state_vec.push_back(std::to_string(exp.get_int_param("run")));
            state_vec.push_back(std::to_string(counter));
            state_vec.push_back(std::to_string(cur_state[0]));
            state_vec.push_back(std::to_string(cur_state[1]));
            state_vec.push_back(std::to_string(cur_state[2]));
            state_vec.push_back(std::to_string(target));
            state_vec.push_back(std::to_string(prediction));
            state_vec.push_back(std::to_string(env.get_L()));
            state_vec.push_back(std::to_string(env.get_seq_length()));
            state_vec.push_back(std::to_string(env.get_data_timestep()));
            state_logger.push_back(state_vec);
        }

        if(counter < 10){
            std::string g = my_network.get_viz_graph();
            std::vector<std::string> graph_data;
            graph_data.push_back(std::to_string(exp.get_int_param("steps")));
            graph_data.push_back(std::to_string(exp.get_int_param("run")));
            graph_data.push_back(g);
            graph_logger.push_back(graph_data);
        }

        if(counter % 50000 < 200)
        {
            std::vector<float> cur_state = env.get_state();
            cur_state.push_back(target);
            cur_state.push_back(prediction);
            cur_state.push_back(env.get_L());
            cur_state.push_back(env.get_data_timestep());
            print_vector(cur_state);
        }

        if(counter%500000  == 499999 && exp.get_bool_param("add_features"))
        {
            my_network.add_memory(exp.get_float_param("step_size"));
            my_network.add_memory(exp.get_float_param("step_size"));
            my_network.add_memory(exp.get_float_param("step_size"));
            my_network.add_memory(exp.get_float_param("step_size"));
            my_network.add_memory(exp.get_float_param("step_size"));

            std::string g = my_network.get_viz_graph();
            std::vector<std::string> graph_data;
            graph_data.push_back(std::to_string(exp.get_int_param("steps")));
            graph_data.push_back(std::to_string(exp.get_int_param("run")));
            graph_data.push_back(g);
            graph_logger.push_back(graph_data);
        }

        if(counter % 100000 == 99998)
        {
//            print_vector(my_network.get_memory_weights());
            std::cout << "Pushing results" << std::endl;
            synapses_metric.add_values(error_logger);
            observations_metric.add_values(state_logger);
            std::cout << "Results added " << std::endl;
            std::cout << "Len = " << error_logger.size() << std::endl;
            error_logger.clear();
            state_logger.clear();
        }
        if (counter % 10000 == 0 || counter % 10000 == 999 || counter % 10000 == 998) {
            std::cout << "### STEP = " << counter << std::endl;
            std::cout << "Running error = " << running_error << std::endl;
        }
    }

    auto end = std::chrono::steady_clock::now();
    std::cout << "Elapsed time in milliseconds for per steps: "
              << 1000000 / (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                            exp.get_int_param("steps"))
              << " fps" << std::endl;


    std::string g = my_network.get_viz_graph();
    std::vector<std::string> graph_data;
    graph_data.push_back(std::to_string(exp.get_int_param("steps")));
    graph_data.push_back(std::to_string(exp.get_int_param("run")));
    graph_data.push_back(g);
    graph_logger.push_back(graph_data);
    graph_state.add_values(graph_logger);

    std::vector<std::string> state_data;
    state_data.push_back(std::to_string(exp.get_int_param("run")));
    state_data.push_back(state);
    state_data.push_back(state_comments);
    run_state_metric.add_value(state_data);

    return 0;
}

//
// Created by Khurram Javed on 2021-04-01.
//

