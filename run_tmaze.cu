#include "hip/hip_runtime.h"
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <map>
#include <string>
#include <signal.h>
#include <random>

#include "include/utils.h"
#include "include/neural_networks/networks/test_network.h"
#include "include/neural_networks/neural_network.h"
#include "include/experiment/Experiment.h"
#include "include/neural_networks/utils.h"
#include "include/experiment/Metric.h"
#include "src/hybrid_code/queue.cu"
#include "include/environments/tmaze.h"
#include "include/animal_learning/tracecondioning.h"
#include "include/neural_networks/networks/test_network.h"

volatile sig_atomic_t someone_killed_me = 0;
void sigint(int sig){
    someone_killed_me = 1;
    std::cout << "\nSIGINT detected, saving results and killing..." << std::endl;
}

int main(int argc, char *argv[]) {
    signal(SIGINT, sigint);

//    TMaze env = TMaze(0, 5);
//    Observation obs = env.reset();
//    std::vector<float> N = {1,0,0,0};
//    std::vector<float> E = {0,1,0,0};
//    std::vector<float> W = {0,0,1,0};
//    std::vector<float> S = {0,0,0,1};
//
//    std::vector<float> direction;
//    std::mt19937 mt(time(0));
//    auto dir_smp = std::uniform_int_distribution<int>(0,3);
//    std::vector<std::tuple<Observation, int, int>> allobs;
//    for (int i = 0; i < 100; i++) {
//        int dir = dir_smp(mt);
//        if (dir == 0)
//            direction = N;
//        else if (dir == 1)
//            direction = E;
//        else if (dir == 2)
//            direction = S;
//        else if (dir == 3)
//            direction = W;
//        allobs.push_back(std::tuple<Observation, int, int>{env.step(direction), dir, env.get_current_pos_in_corridor()});
//    }
//    __builtin_trap();
//
//   TODO it seems to work as intended but should check again probably
//   set print pretty on
//   set $i=0
//   p allobs[$i++]

    std::cout << "Program started \n";
    Experiment exp = Experiment(argc, argv);
    std::cout << "Experiment object created \n";
    int width = exp.get_int_param("width");

    Metric run_state_metric = Metric(exp.database_name, "run_states",
                                     std::vector<std::string>{"run", "state", "state_comments"},
                                     std::vector<std::string>{"int", "VARCHAR(10)", "VARCHAR(30)"},
                                     std::vector<std::string>{"run"});
    Metric observations_metric = Metric(exp.database_name, "run_metrics",
                                        std::vector<std::string>{"run", "step", "episode", "eps_step", "avg_reward",
                                                                 "corridor_len", "corridor_pos", "state", "qvalues",
                                                                 "reward", "new_features"},
                                        std::vector<std::string>{"int", "int", "int", "int", "real", "int", "int",
                                                                 "JSON", "JSON", "real", "int"},
                                        std::vector<std::string>{"run", "step"});
    Metric graph_state = Metric(exp.database_name, "network_graphs",
                                std::vector<std::string>{"run", "step", "graph_data"},
                                std::vector<std::string>{"int", "int", "MEDIUMTEXT"},
                                std::vector<std::string>{"run", "step"});

    //TODO add num inp and out as params here fixed for env
    CustomNetwork my_network = CustomNetwork(exp.get_float_param("step_size"),
                                             exp.get_int_param("width"),
                                             exp.get_int_param("num_layers"),
                                             exp.get_int_param("sparsity"),
                                             exp.get_int_param("seed"));

    TMaze env = TMaze(exp.get_int_param("seed"),
                      exp.get_int_param("tmaze_corridor_length"));

    //get a sequence of data for data-driven initialization
//    if (exp.get_bool_param("data_driven_initialization")){
//        std::vector<std::vector<float>> input_batch;
//        input_batch.reserve(500);
//        for(int temp=0; temp<500; temp++)
//            input_batch.push_back(env.step(1));
//        my_network.initialize_network(input_batch);
//        env.reset();
//    }

    std::cout << "Total synapses in the network " << my_network.get_total_synapses() << std::endl;
    auto start = std::chrono::steady_clock::now();

    float R = 0;
    float R_old = 0;
    float average_reward = 0;
    float accuracy = -1;
    int selected_action_idx_old = 0;
    float gamma = exp.get_float_param("gamma");
    bool prev_was_terminal = false;

    int timestep_since_feat_added = exp.get_int_param("features_min_timesteps");
    int total_new_features = 0;
    std::string state = "finished";
    std::string state_comments = "";
    std::vector<std::vector<std::string>> metric_logger;
    std::vector<std::vector<std::string>> graph_logger;
    std::mt19937 mt(exp.get_int_param("seed"));
    auto exploration_sampler = std::uniform_int_distribution<int>(0,100);
    auto rnd_action_sampler = std::uniform_int_distribution<int>(0,3);

    for (int counter = 0; counter < exp.get_int_param("steps"); counter++) {
        if(someone_killed_me){
            state = "killed";
            state_comments = "interrupt_sig";
            break;
        }

        //TODO consider whether we want to add feats from start
        timestep_since_feat_added -= 1;

        Observation current_obs = env.get_current_obs();
        R_old = R;
        R = current_obs.reward;

        my_network.set_input_values(current_obs.state);
        my_network.step();
        //TODO env step here
        std::vector<float> qvalues = my_network.read_output_values();
        std::vector<float> action(qvalues.size(), 0.0);
        std::vector<float> targets(qvalues.size(), 0.0);
        int selected_action_idx = 0;
        if (exploration_sampler(mt) < exp.get_float_param("epsilon")*100)
            selected_action_idx = rnd_action_sampler(mt);
        else
            selected_action_idx = std::distance(qvalues.begin(), std::max_element(qvalues.begin(), qvalues.end()));
        action[selected_action_idx] = 1;
        //update the gradient for only the old action since current one is for bootstrap
        if (prev_was_terminal){
            // if previous state was terminal state, we dont want next episode's values to propagate into it
            targets[selected_action_idx_old] = R_old;
            prev_was_terminal = false;
        }
        else
            targets[selected_action_idx_old] = R_old + gamma * qvalues[selected_action_idx];
        selected_action_idx_old = selected_action_idx;

        if (counter > 0){
            my_network.introduce_targets(targets);
            average_reward = 0.999 * average_reward + 0.001 * R;
            if (current_obs.is_terminal){
                prev_was_terminal = true;
                if (accuracy == -1)
                    accuracy = int(R==4);
                else
                    accuracy = 0.999 * accuracy + 0.001 * int(R==4);
            }
        }
        if(counter % 50000 < 50000)
        {
            std::vector<float> cur_state = current_obs.state;
            cur_state.push_back(current_obs.episode);
            cur_state.push_back(current_obs.timestep);
            cur_state.push_back(current_obs.reward);
            cur_state.push_back(average_reward);
            cur_state.push_back(accuracy);
            cur_state.push_back(env.get_current_pos_in_corridor());
            print_vector(cur_state);
            print_vector(qvalues);
        }

        //TODO the qvalues used to make this action belong to old state
        current_obs = env.step(action);

        if(isnan(qvalues[selected_action_idx])){
          state = "killed";
          state_comments = "nan_prediction";
          std::cout << "killing due to nans" << std::endl;
          break;
        }

        if(counter < 10){
            std::string g = my_network.get_viz_graph();
            std::vector<std::string> graph_data;
            graph_data.push_back(std::to_string(counter));
            graph_data.push_back(std::to_string(exp.get_int_param("run")));
            graph_data.push_back(g);
            graph_logger.push_back(graph_data);
        }


        if(exp.get_bool_param("add_features") &&
           timestep_since_feat_added < 1)
        {
            total_new_features += exp.get_int_param("num_new_features");
            timestep_since_feat_added = exp.get_int_param("features_min_timesteps");
            for (int i = 0; i < exp.get_int_param("num_new_features"); i++)
                my_network.add_memory(exp.get_float_param("step_size"));

            std::cout << "\n Adding features..." << std::endl;

            std::string g = my_network.get_viz_graph();
            std::vector<std::string> graph_data;
            graph_data.push_back(std::to_string(counter));
            graph_data.push_back(std::to_string(exp.get_int_param("run")));
            graph_data.push_back(g);
            graph_logger.push_back(graph_data);
        }

//        if(counter % 300000 == 299998)
//        {
//            if(exp.get_bool_param("add_features"))
//                print_vector(my_network.get_memory_weights());
//            std::cout << "Pushing results" << std::endl;
//            observations_metric.add_values(obs_logger);
//            graph_state.add_values(graph_logger);
//            std::cout << "Results added " << std::endl;
//            std::cout << "Len = " << error_logger.size() << std::endl;
//            error_logger.clear();
//            obs_logger.clear();
//            graph_logger.clear();
//        }
//        if (counter % 10000 == 0 || counter % 10000 == 999 || counter % 10000 == 998) {
//            std::cout << "### STEP = " << counter << std::endl;
//            std::cout << "Running error = " << running_error << std::endl;
//            std::cout << "Running accuracy = " << running_accuracy << std::endl;
//        }
    }

    auto end = std::chrono::steady_clock::now();
    std::cout << "Elapsed time in milliseconds for per steps: "
              << 1000000 / (1+(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                            exp.get_int_param("steps")))
              << " fps" << std::endl;

    //observations_metric.add_values(obs_logger);
    std::string g = my_network.get_viz_graph();
    std::vector<std::string> graph_data;
    std::cout << g << std::endl;
    graph_data.push_back(std::to_string(exp.get_int_param("steps")));
    graph_data.push_back(std::to_string(exp.get_int_param("run")));
    graph_data.push_back(g);
    graph_logger.push_back(graph_data);
    graph_state.add_values(graph_logger);

    std::vector<std::string> state_data;
    state_data.push_back(std::to_string(exp.get_int_param("run")));
    state_data.push_back(state);
    state_data.push_back(state_comments);
    run_state_metric.add_value(state_data);

    return 0;
}
